/*
 * Copyright (C) 2016 Takuya KOUMURA
 * https://github.com/takuya-koumura/birdsong-recognition
 *
 * This file is part of Birdsong Recognition.
 * 
 * Birdsong Recognition is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * Birdsong Recognition is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with Birdsong Recognition.  If not, see <http://www.gnu.org/licenses/>.
 */


#include <hip/hip_runtime.h>

#include <cmath>

extern "C" __global__ void SeqSoftmaxConvBackwardDoubleChar(const char *label, double *diff, const int *size, int labelShiftY, int labelShiftX)
{
	int batchSize = size[0];
	int numChannel= size[1];
	int height = size[2];
	int width = size[3];
	int singleSize= size[4];
	int singleHeight= size[5];
	int singleWidth= size[6];
	int labelHeight = size[7];
	int labelWidth = size[8];
	int labelShiftUpperH = size[9];
	int labelShiftUpperW = size[10];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batchSize*height*width*singleHeight*singleWidth) return;

	int x = idx%width;
	int i = (idx - x) / width;
	int y = i%height;
	i = (idx - y) / height;
	int sx = i%singleWidth;
	i = (i - sx) / singleWidth;
	int sy = i%singleHeight;
	int b = i / singleHeight;

	char la = label[b*labelWidth*labelHeight + (y*labelShiftUpperH + labelShiftY + sy)*labelWidth + x*labelShiftUpperW + labelShiftX + sx];

	if (la < 0)
	{
		for (int li = 0; li < singleSize; ++li)
		{
			int index = ((b*numChannel + (li*singleHeight + sy)*singleWidth + sx)*height + y)*width + x;
			diff[index] = 0;
		}
		return;
	}
	int index = ((b*numChannel + (la*singleHeight + sy)*singleWidth + sx)*height + y)*width + x;
	diff[index] -= 1;
}

extern "C" __global__ void SeqSoftmaxConvBackwardFloatChar(const char *label, float *diff, const int *size, int labelShiftY, int labelShiftX)
{
	int batchSize = size[0];
	int numChannel = size[1];
	int height = size[2];
	int width = size[3];
	int singleSize = size[4];
	int singleHeight = size[5];
	int singleWidth = size[6];
	int labelHeight = size[7];
	int labelWidth = size[8];
	int labelShiftUpperH = size[9];
	int labelShiftUpperW = size[10];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batchSize*height*width*singleHeight*singleWidth) return;

	int x = idx%width;
	int i = (idx - x) / width;
	int y = i%height;
	i = (idx - y) / height;
	int sx = i%singleWidth;
	i = (i - sx) / singleWidth;
	int sy = i%singleHeight;
	int b = i / singleHeight;

	char la = label[b*labelWidth*labelHeight + (y*labelShiftUpperH + labelShiftY + sy)*labelWidth + x*labelShiftUpperW + labelShiftX + sx];

	if (la < 0)
	{
		for (int li = 0; li < singleSize; ++li)
		{
			int index = ((b*numChannel + (li*singleHeight + sy)*singleWidth + sx)*height + y)*width + x;
			diff[index] = 0;
		}
		return;
	}
	int index = ((b*numChannel + (la*singleHeight + sy)*singleWidth + sx)*height + y)*width + x;
	diff[index] -= 1;
}


extern "C" __global__ void SeqSoftmaxConvErrorDoubleChar(double *error, const double *output, const char *label, const int *size, int labelShiftY, int labelShiftX, double outputLowerForError)
{
	int batchSize = size[0];
	int numChannel = size[1];
	int height = size[2];
	int width = size[3];
	int singleSize = size[4];
	int singleHeight = size[5];
	int singleWidth = size[6];
	int labelHeight = size[7];
	int labelWidth = size[8];
	int labelShiftUpperH = size[9];
	int labelShiftUpperW = size[10];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batchSize*height*width*singleHeight*singleWidth) return;

	int x = idx%width;
	int i = (idx - x) / width;
	int y = i%height;
	i = (idx - y) / height;
	int sx = i%singleWidth;
	i = (i - sx) / singleWidth;
	int sy = i%singleHeight;
	int b = i / singleHeight;

	int labelIndex = b*labelWidth*labelHeight + (y*labelShiftUpperH + labelShiftY + sy)*labelWidth + x*labelShiftUpperW + labelShiftX + sx;
	char la = label[labelIndex];

	if (la < 0)
	{
		error[idx] = 0;
		return;
	}

	int index = ((b*numChannel + (la*singleHeight + sy)*singleWidth + sx)*height + y)*width + x;
	double o = output[index];
	if (isnan(o)) error[idx] = o;
	else
	{
		if (o < outputLowerForError) o = outputLowerForError;
		error[idx] = -log(o);
	}
}


extern "C" __global__ void SeqSoftmaxConvErrorFloatChar(float *error, const float *output, const char *label, const int *size, int labelShiftY, int labelShiftX, float outputLowerForError)
{
	int batchSize = size[0];
	int numChannel = size[1];
	int height = size[2];
	int width = size[3];
	int singleSize = size[4];
	int singleHeight = size[5];
	int singleWidth = size[6];
	int labelHeight = size[7];
	int labelWidth = size[8];
	int labelShiftUpperH = size[9];
	int labelShiftUpperW = size[10];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= batchSize*height*width*singleHeight*singleWidth) return;

	int x = idx%width;
	int i = (idx - x) / width;
	int y = i%height;
	i = (idx - y) / height;
	int sx = i%singleWidth;
	i = (i - sx) / singleWidth;
	int sy = i%singleHeight;
	int b = i / singleHeight;

	int labelIndex = b*labelWidth*labelHeight + (y*labelShiftUpperH + labelShiftY + sy)*labelWidth + x*labelShiftUpperW + labelShiftX + sx;
	char la = label[labelIndex];

	if (la < 0)
	{
		error[idx] = 0;
		return;
	}

	int index = ((b*numChannel + (la*singleHeight + sy)*singleWidth + sx)*height + y)*width + x;
	float o = output[index];
	if (isnan(o)) error[idx] = o;
	else
	{
		if (o < outputLowerForError) o = outputLowerForError;
		error[idx] = -log(o);
	}
}


extern "C" __global__ void FillFloat(float *vector, float value, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size) return;
	
	vector[idx] = value;
}

extern "C" __global__ void FillDouble(double *vector, double value, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size) return;

	vector[idx] = value;
}


extern "C" __global__ void AdamFloat(float *param, const float* grad, float* moment, float* moment2, const float* hyperParam, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size) return;

	float alpha = hyperParam[0];
	float beta1= hyperParam[1];
	float beta2 = hyperParam[2];
	float epsilon = hyperParam[3];
	float beta1T = hyperParam[4];
	float beta2T = hyperParam[5];

	moment[idx] = beta1*moment[idx] + (1 - beta1)*grad[idx];
	moment2[idx] = beta2*moment2[idx] + (1 - beta2)*grad[idx] * grad[idx];
	float alphaT = alpha*sqrt(1 - beta2T) / (1 - beta1T);
	float delta = alphaT*moment[idx] / (sqrt(moment2[idx]) + epsilon);
	param[idx] -= delta;
}

extern "C" __global__ void AdamDouble(double *param, const double* grad, double* moment, double* moment2, const double* hyperParam, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size) return;

	double alpha = hyperParam[0];
	double beta1 = hyperParam[1];
	double beta2 = hyperParam[2];
	double epsilon = hyperParam[3];
	double beta1T = hyperParam[4];
	double beta2T = hyperParam[5];

	moment[idx] = beta1*moment[idx] + (1 - beta1)*grad[idx];
	moment2[idx] = beta2*moment2[idx] + (1 - beta2)*grad[idx] * grad[idx];
	double alphaT = alpha*sqrt(1 - beta2T) / (1 - beta1T);
	double delta = alphaT*moment[idx] / (sqrt(moment2[idx]) + epsilon);
	param[idx] -= delta;
}
